#include "hip/hip_runtime.h"
/*
 **********************************************
 *  CS314 Principles of Programming Languages *
 *  Fall 2020                                 *
 **********************************************
 */
#include "utils.hpp"
#include "gpuHeaders.cuh"
#include "extra.cu"
#include <iostream>

using namespace std;

#define threadsPerBlock 256

int one_way_handshake(GraphData graph, int *& matches, int numthreads, bool extra_credit)
{
	int num_thread_blocks = (numthreads + threadsPerBlock - 1) / threadsPerBlock;
	
	int numVertices = graph.numNodes;
	int numEdges = graph.numEdges;
	
	//Prepare various GPU arrays that we're going to need:
	int * strongNeighbor_gpu;//will hold strongest neighbor for each vertex
	int * matches_gpu;//will hold the output
	int * src_gpu;//holds the src nodes in edge list
	int * dst_gpu;//holds the dst nodes in edge list
	int * weight_gpu;//holds the edge weights in edge list	
	int * temp1_gpu;//a temporary array for data we don't need to keep for long
	int * temp2_gpu;//a temporary array for data we don't need to keep for long
	int * temp3_gpu;//a temporary array for data we don't need to keep for long
	int * temp4_gpu;//a temporary array for data we don't need to keep for long
	
	/** YOUR CODE GOES BELOW (allocate GPU memory, and copy from CPU to GPU as appropriate **/
	hipMalloc((void **)&strongNeighbor_gpu, sizeof(int)*numVertices);

	hipMalloc((void **)&matches_gpu, sizeof(int)*numVertices);
	int * matches_cpu = (int *)malloc(sizeof(int) * numVertices);
	for(int i = 0; i < numVertices; i++){
		matches_cpu[i] = -1;
	}
	hipMemcpy(matches_gpu, matches_cpu, sizeof(int) * numVertices, hipMemcpyHostToDevice);
	free(matches_cpu);
	hipMalloc((void **)&src_gpu, sizeof(int) * (numEdges));
	hipMemcpy(src_gpu, graph.src, sizeof(int) * (numEdges), hipMemcpyHostToDevice);
	
	hipMalloc((void **)&dst_gpu, sizeof(int) * (numEdges));
	hipMemcpy(dst_gpu, graph.dst, sizeof(int) * (numEdges), hipMemcpyHostToDevice);
	
	hipMalloc((void **)&weight_gpu, sizeof(int) * (numEdges));
	hipMemcpy(weight_gpu, graph.weight, sizeof(int) * (numEdges), hipMemcpyHostToDevice);

	hipMalloc((void **)&temp1_gpu, sizeof(int) * (numEdges+1));
	
	hipMalloc((void **)&temp2_gpu, sizeof(int) * (numEdges+1));

	hipMalloc((void **)&temp3_gpu, sizeof(int) * (numEdges+1));

	hipMalloc((void **)&temp4_gpu, sizeof(int) * (numEdges+1));

	/** YOUR CODE GOES ABOVE **/



    //matching loop
    int iter;
    for (iter = 0; ; iter++) {
		
		if(extra_credit) {
			/** YOUR CODE GOES BELOW (extra credit) **/

			/** YOUR CODE GOES ABOVE (extra credit) **/
		} else {
			//Step 1: Get strongest neighbor for each vertex/node
			int * strongNeighbor_cpu = (int *) malloc(sizeof(int) * numVertices);
			int * strongNeighborWeight_cpu = (int *) malloc(sizeof(int) * numVertices);
			for(int x = 0; x < numVertices; x++) {
				strongNeighbor_cpu[x] = -1;
			}
			for(int x = 0; x < numEdges; x++) {
				int src = graph.src[x];
				int dst = graph.dst[x];
				int wgt = graph.weight[x];
				//std::cerr << src << "->" << dst << ": " << wgt << "\n";
				if(strongNeighbor_cpu[src] == -1 || strongNeighborWeight_cpu[src] < wgt) {
					strongNeighbor_cpu[src] = dst;
					strongNeighborWeight_cpu[src] = wgt;
				}
			}
			
			//move data from CPU to GPU, and free the CPU arrays
			hipMemcpy(strongNeighbor_gpu, strongNeighbor_cpu, numVertices * sizeof(int), hipMemcpyHostToDevice);
			free(strongNeighbor_cpu);
			free(strongNeighborWeight_cpu);
		}
		
		//Step 2: check for each vertex whether there's a handshake
		check_handshaking_gpu<<<num_thread_blocks, threadsPerBlock>>>(strongNeighbor_gpu, matches_gpu, numVertices);
		
		//Step 3: filter
		
		//Step 3a: decide which edges to keep (marked with a 1) versus filter (marked with a 0)
		int * keepEdges_gpu = temp1_gpu;
		temp1_gpu = NULL;
		markFilterEdges_gpu<<<num_thread_blocks, threadsPerBlock>>>(src_gpu, dst_gpu, matches_gpu, keepEdges_gpu, numEdges);
		
		
		//Step 3b: get new indices (in edge list for next iteration) of the edges we're going to keep
		int * newEdgeLocs_gpu = keepEdges_gpu;
		keepEdges_gpu = NULL;
		for(int distance = 0; distance <= numEdges; distance = max(1, distance * 2)) {
			exclusive_prefix_sum_gpu<<<num_thread_blocks, threadsPerBlock>>>(newEdgeLocs_gpu, temp2_gpu, distance, numEdges+1);
			swapArray((void**) &newEdgeLocs_gpu, (void**) &temp2_gpu);
		}
		
		//note: temp1 is still in use, until we're done with newEdgeLocs_gpu
		
		//Step 3c: check if we're done matching
		int lastLoc = 0;
		hipMemcpy(&lastLoc, &(newEdgeLocs_gpu[numEdges]), sizeof(int), hipMemcpyDeviceToHost);
		if(lastLoc < 2) {
			//termination: fewer than two nodes remain unmatched
			break;
		} else if(lastLoc == numEdges) {
			//termination: no additional matches are possible
			break;
		}
		
		//Step 3d: pack the src, dst, and weight arrays in accordance with new edge locations
		packGraph_gpu<<<num_thread_blocks, threadsPerBlock>>>(temp2_gpu, src_gpu, temp3_gpu, dst_gpu, temp4_gpu, weight_gpu, newEdgeLocs_gpu, numEdges);
		swapArray((void**) &temp2_gpu, (void**) &src_gpu);
		swapArray((void**) &temp3_gpu, (void**) &dst_gpu);
		swapArray((void**) &temp4_gpu, (void**) &weight_gpu);
		
		temp1_gpu = newEdgeLocs_gpu;
		newEdgeLocs_gpu = NULL;
		
		//note: now we're done with the current contents of all the temporary arrays
		
		//Set new number of edges:
		numEdges = lastLoc;
		
		if(iter > numVertices) {
			cerr << "Error: matching has been running too long; breaking loop now\n";
			break;
		}
		
		if(!extra_credit) {
			//Step 4: Copy new graph arrays to CPU
			hipMemcpy(graph.src, src_gpu, numEdges * sizeof(int), hipMemcpyDeviceToHost);
			hipMemcpy(graph.dst, dst_gpu, numEdges * sizeof(int), hipMemcpyDeviceToHost);
			hipMemcpy(graph.weight, weight_gpu, numEdges * sizeof(int), hipMemcpyDeviceToHost);
		}
    }
	
	hipMemcpy(matches, matches_gpu, numVertices * sizeof(int), hipMemcpyDeviceToHost);
	
	//Wait until pending GPU operations are complete:
	hipDeviceSynchronize();
	
	//free GPU arrays
	/** YOUR CODE GOES BELOW **/

	hipFree(strongNeighbor_gpu);
	hipFree(matches_gpu);
	hipFree(src_gpu);
	hipFree(dst_gpu);
	hipFree(weight_gpu);
	hipFree(temp1_gpu);
	hipFree(temp2_gpu);
	hipFree(temp3_gpu);
	hipFree(temp4_gpu);

	/** YOUR CODE GOES ABOVE **/
	
	hipError_t hipError_t;
	hipError_t = hipGetLastError();
	if(hipError_t != hipSuccess) {
		cerr << "Warning: one or more CUDA errors occurred. Try using cuda-gdb to debug. Error message: \n\t" <<hipGetErrorString(hipError_t) << "\n";
	}
	
	return iter + 1;
}

void one_way_handshake_wrapper(GraphData graph, int *& matches, int numthreads, bool extra_credit)
{
	fprintf(stderr, "Start One Way Matching ... \n");

    struct timeval beginTime, endTime;

    setTime(&beginTime);

	int iter = one_way_handshake(graph, matches, numthreads, extra_credit);

    setTime(&endTime);

    fprintf(stderr, "Done matching.\n");

    fprintf(stderr, "Performed matching for %d iterations\n", iter);
    fprintf(stderr, "One Way Handshaking Matching Time: %.2f ms\n",
            getTime(&beginTime, &endTime));
}
