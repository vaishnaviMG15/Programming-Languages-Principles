/*
 **********************************************
 *  CS314 Principles of Programming Languages *
 *  Fall 2020                                 *
 **********************************************
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void exclusive_prefix_sum_gpu(int * oldSum, int * newSum, int distance, int numElements) {
	/** YOUR CODE GOES BELOW **/

	
	int totalThreads = blockDim.x * gridDim.x;
	int tid = (blockDim.x * blockIdx.x) + threadIdx.x;

	if (totalThreads >= numElements){

		if(distance == 0){
			if (tid == 0){
				newSum[tid] = 0;
			}else if (tid < numElements){
				newSum[tid] = oldSum[tid - 1];
			}
		}else{
			if((tid >= distance) && (tid < numElements)){
					
				newSum[tid] = oldSum[tid] + oldSum[tid - distance];

			}else if (tid < distance){
				newSum[tid] = oldSum[tid];
			}
		}
	}else{
		for(int i = tid; i < numElements; i += totalThreads){

			if(distance == 0){
				if (i == 0){
					newSum[i] = 0;
				}else{
					newSum[i] = oldSum[i - 1];
				}
			}else{
				if(i >= distance){
				
					newSum[i] = oldSum[i] + oldSum[i - distance];

				}else if (i < distance){
					newSum[i] = oldSum[i];
				}
			}

		}
	}


	/** YOUR CODE GOES ABOVE **/
}
