/*
 **********************************************
 *  CS314 Principles of Programming Languages *
 *  Fall 2020                                 *
 **********************************************
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void markFilterEdges_gpu(int * src, int * dst, int * matches, int * keepEdges, int numEdges) {
	/** YOUR CODE GOES BELOW **/
	

	int totalThreads = blockDim.x * gridDim.x;
	int tid = (blockDim.x * blockIdx.x) + threadIdx.x;

	if(totalThreads >= numEdges){

		if (tid < numEdges){

			if((matches[src[tid]] == -1) && (matches[dst[tid]] == -1)){

				keepEdges[tid] = 1;

			}else{

				keepEdges[tid] = 0;
			}
		}
	}else{

		for (int i = tid; i < numEdges; i += totalThreads){


				if((matches[src[i]] == -1) && (matches[dst[i]] == -1)){

					keepEdges[i] = 1;

				}else{

					keepEdges[i] = 0;
				}

		}	
	}



	/** YOUR CODE GOES ABOVE **/
}
