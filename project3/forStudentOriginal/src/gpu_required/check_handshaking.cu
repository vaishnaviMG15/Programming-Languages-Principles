/*
 **********************************************
 *  CS314 Principles of Programming Languages *
 *  Fall 2020                                 *
 **********************************************
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void check_handshaking_gpu(int * strongNeighbor, int * matches, int numNodes) {
	/** YOUR CODE GOES BELOW **/
	
		int totalThreads = blockDim.x * gridDim.x;
		int tid = (blockDim.x * blockIdx.x) + threadIdx.x;

		if (totalThreads >= numNodes){
			
			if(tid < numNodes){

				if(strongNeighbor[tid] != -1){

					if(strongNeighbor[strongNeighbor[tid]] == tid){

						matches[tid] = strongNeighbor[tid];

					}else{

						matches[tid] = -1;

					}
					

				}
					
			}	

		}else{

			for(int i = tid; i < numNodes; i += totalThreads){
				
				
				if(strongNeighbor[i] != -1){

					if(strongNeighbor[strongNeighbor[i]] == i){

						matches[i] = strongNeighbor[i];

					}else{

						matches[i] = -1;

					}
					

				}

			}

		}



	/** YOUR CODE GOES ABOVE **/
}
