/*
 **********************************************
 *  CS314 Principles of Programming Languages *
 *  Fall 2020                                 *
 **********************************************
 */
#include "utils.hpp"
#include <strings.h>
#include "DataStructure.hpp"
#include "onewaywrapper.hpp"
#include <iostream>
using namespace std;

int main(int argc, char *argv[])
{
	char * filenameGraph = NULL;
	char * filenameOutput = (char *) "out.txt";
	int CUDA_device = 0;
	int num_threads = 16384;
	bool extra_credit = false;
	
	//examine command-line arguments:
    for(int arg = 1; arg < argc; arg++) {
		if(!strcmp(argv[arg], "-device")) {
			arg++;
			if(arg >= argc) {//bad arguments
				filenameGraph = NULL;
				break;
			}
			CUDA_device = atoi(argv[arg]);
		} else if(!strcmp(argv[arg], "-input")) {
			arg++;
			if(arg >= argc) {//bad arguments
				filenameGraph = NULL;
				break;
			}
			filenameGraph = argv[arg];
		} else if(!strcmp(argv[arg], "-output")) {
			arg++;
			if(arg >= argc) {//bad arguments
				filenameGraph = NULL;
				break;
			}
			filenameOutput = argv[arg];
		} else if(!strcmp(argv[arg], "-threads")) {
			arg++;
			if(arg >= argc) {//bad arguments
				filenameGraph = NULL;
				break;
			}
			num_threads = atoi(argv[arg]);
		} else if(!strcmp(argv[arg], "-extra")) {
			extra_credit = true;
		} else {
			filenameGraph = NULL;
			break;
		}
	}
	
	//if given invalid arguments, print usage info and quit:
	if (filenameGraph == NULL) {
        printUsage(argv[0]);
        exit(EXIT_FAILURE);
    }
	
	//set CUDA device to specified GPU:
	hipSetDevice(CUDA_device);
	
	GraphData graph;
	
    //read the matrix/graph from the matrix market format file(.mtx) and sort it
    readmm(filenameGraph, &graph);

    //allocate memory for matching result
    int * res = (int *) malloc(graph.numNodes * sizeof(int));
	
    //initialize res to UNMATCHED
    for (int i = 0 ; i < graph.numNodes; i++) res[i] = -1;
	
    one_way_handshake_wrapper(graph, res, num_threads, extra_credit);
	
    //write result to output file
    write_match_result(filenameOutput, res, graph.numNodes);

    //clean allocated memory
	free(res);
	free(graph.src);
	free(graph.dst);
	free(graph.weight);

    return 0;
}
