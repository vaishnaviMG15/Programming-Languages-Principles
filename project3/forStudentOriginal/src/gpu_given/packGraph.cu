/*
 **********************************************
 *  CS314 Principles of Programming Languages *
 *  Fall 2020                                 *
 **********************************************
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void packGraph_gpu(int * newSrc, int * oldSrc, int * newDst, int * oldDst, int * newWeight, int * oldWeight, int * edgeMap, int numEdges) {
	//Get current thread ID and total number of threads in grid
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int num_threads = blockDim.x * gridDim.x;
	
	for(int x = tid; x < numEdges; x += num_threads) {
		int myMap = edgeMap[x];
		int nextMap = edgeMap[x + 1];
		
		if(myMap != nextMap) {
			newSrc[myMap] = oldSrc[x];
			newDst[myMap] = oldDst[x];
			newWeight[myMap] = oldWeight[x];
		}
	}
}
